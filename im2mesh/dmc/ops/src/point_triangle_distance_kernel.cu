#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>


__constant__ float eps=1e-8;

__constant__ float grid_size = 1.0;

__constant__ float distance_empty = 0.4;

// first row, topology
// second row, number of triangles in the corresponding topology
// up to __3__ triangles
__constant__ int acceptTopology[2][48] = {{1, 2, 3, 4, 6, 7, 8, 9, 11, 12, 13, 14, 15, 16, 17, 19, 25, 31, 32, 34, 35, 38, 47, 48, 49, 50, 51, 55, 59, 63, 64, 68, 70, 76, 79, 96, 98, 100, 102, 103, 110, 111, 112, 115, 118, 119, 127, 0},
			{1, 1, 2, 1, 2, 3, 1, 2, 3, 2, 3, 3, 2, 1, 2, 3, 3, 3, 1, 2, 3, 3, 3, 2, 3, 3, 2, 3, 3, 2, 1, 2, 3, 3, 3, 2, 3, 3, 2, 3, 3, 2, 3, 3, 3, 2, 1, 0}};


__constant__ int triTable[256][16] =
{{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1},
{3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1},
{3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1},
{3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1},
{9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1},
{9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
{2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1},
{8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1},
{9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
{4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1},
{3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1},
{1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1},
{4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1},
{4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
{5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1},
{2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1},
{9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
{0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
{2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1},
{10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1},
{5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1},
{5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1},
{9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1},
{0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1},
{1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1},
{10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1},
{8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1},
{2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1},
{7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1},
{2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1},
{11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1},
{5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1},
{11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1},
{11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
{1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1},
{9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1},
{5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1},
{2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
{5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1},
{6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1},
{3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1},
{6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1},
{5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1},
{1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
{10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1},
{6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1},
{8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1},
{7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1},
{3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
{5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1},
{0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1},
{9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1},
{8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1},
{5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1},
{0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1},
{6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1},
{10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1},
{10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1},
{8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1},
{1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1},
{0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1},
{10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1},
{3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1},
{6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1},
{9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1},
{8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1},
{3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1},
{6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1},
{0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1},
{10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1},
{10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1},
{2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1},
{7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1},
{7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1},
{2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1},
{1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1},
{11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1},
{8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1},
{0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1},
{7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
{10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
{2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
{6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1},
{7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1},
{2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1},
{1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1},
{10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1},
{10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1},
{0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1},
{7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1},
{6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1},
{8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1},
{9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1},
{6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1},
{4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1},
{10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1},
{8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1},
{0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1},
{1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1},
{8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1},
{10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1},
{4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1},
{10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
{5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
{11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1},
{9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
{6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1},
{7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1},
{3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1},
{7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1},
{3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1},
{6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1},
{9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1},
{1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1},
{4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1},
{7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1},
{6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1},
{3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1},
{0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1},
{6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1},
{0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1},
{11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1},
{6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1},
{5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1},
{9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1},
{1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1},
{1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1},
{10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1},
{0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1},
{5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1},
{10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1},
{11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1},
{9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1},
{7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1},
{2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1},
{8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1},
{9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1},
{9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1},
{1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1},
{9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1},
{9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1},
{5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1},
{0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1},
{10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1},
{2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1},
{0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1},
{0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1},
{9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1},
{5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1},
{3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1},
{5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1},
{8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1},
{0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1},
{9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1},
{1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1},
{3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1},
{4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1},
{9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1},
{11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1},
{11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1},
{2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1},
{9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1},
{3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1},
{1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1},
{4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1},
{3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1},
{0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1},
{9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1},
{1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1}};

__constant__ int vertices_to_offset[12][4]={ {0, 1, 1, 0}, // #0
				{1, 1, 1, 0}, // #1
				{0, 1, 0, 0}, // #2
				{1, 0, 1, 0}, // #3
				
				{0, 1, 1, 1}, // #4
				{1, 1, 1, 1}, // #5
				{0, 1, 0, 1}, // #6
				{1, 0, 1, 1}, // #7

				{2, 0, 1, 1}, // #8
				{2, 1, 1, 1}, // #9
				{2, 1, 0, 1}, // #10
				{2, 0, 0, 1}}; // #11

        namespace{
/**
 * convert vertex displacement field to vertices locations
 * params:
 * 	offset		input, vertex displacement field, 3xWxHxD
 * 	W 		input, number of cells on one of the directions
 * 	H 		input, number of cells on one of the directions
 * 	D 		input, number of cells on one of the directions
 * 	x 		input, indice of a cell in the full grid on one of the directions
 * 	y 		input, indice of a cell in the full grid on one of the directions
 * 	z 		input, indice of a cell in the full grid on one of the directions
 * 	vertices 	output, the location of 12 vertices for the specific cell, 3x12 
 *
 */
 template <typename scalar_t>
__device__ void offset_to_vertices_cuda(const scalar_t *offset, const int W, const int H, const int D, const int x, const int y, const int z, scalar_t *vertices){
  // #0
  vertices[0 ] = 0.5-offset[0       + (x+1)*H*D + (y+1)*D + z   ]; 
  vertices[1 ] = 1.0; 
  vertices[2 ] = 0.0; 
  // #1
  vertices[3 ] = 1.0; 
  vertices[4 ] = 0.5-offset[1*W*H*D + (x+1)*H*D + (y+1)*D + z   ]; 
  vertices[5 ] = 0.0; 
  // #2
  vertices[6 ] = 0.5-offset[0       + (x+1)*H*D + (y  )*D + z   ]; 
  vertices[7 ] = 0.0; 
  vertices[8 ] = 0.0; 
  // #3
  vertices[9 ] = 0.0; 
  vertices[10] = 0.5-offset[1*W*H*D + (x  )*H*D + (y+1)*D + z   ]; 
  vertices[11] = 0.0; 

  // #4
  vertices[12] = 0.5-offset[0       + (x+1)*H*D + (y+1)*D + z+1 ]; 
  vertices[13] = 1.0; 
  vertices[14] = 1.0; 
  // #5
  vertices[15] = 1.0; 
  vertices[16] = 0.5-offset[1*W*H*D + (x+1)*H*D + (y+1)*D + z+1 ]; 
  vertices[17] = 1.0; 
  // #6
  vertices[18] = 0.5-offset[0       + (x+1)*H*D + (y  )*D + z+1 ]; 
  vertices[19] = 0.0; 
  vertices[20] = 1.0; 
  // #7
  vertices[21] = 0.0; 
  vertices[22] = 0.5-offset[1*W*H*D + (x  )*H*D + (y+1)*D + z+1 ]; 
  vertices[23] = 1.0; 

  // #8
  vertices[24] = 0.0; 
  vertices[25] = 1.0; 
  vertices[26] = 0.5-offset[2*W*H*D + (x  )*H*D + (y+1)*D + z+1 ]; 
  // #9
  vertices[27] = 1.0; 
  vertices[28] = 1.0; 
  vertices[29] = 0.5-offset[2*W*H*D + (x+1)*H*D + (y+1)*D + z+1 ]; 
  // #10
  vertices[30] = 1.0; 
  vertices[31] = 0.0; 
  vertices[32] = 0.5-offset[2*W*H*D + (x+1)*H*D + (y  )*D + z+1 ]; 
  // #11
  vertices[33] = 0.0; 
  vertices[34] = 0.0; 
  vertices[35] = 0.5-offset[2*W*H*D + (x  )*H*D + (y  )*D + z+1 ]; 
}

/**
 * d_sqrdistance/d_x
 */
 template <typename scalar_t>
__device__ scalar_t d_sqrdistance_(scalar_t a, scalar_t b, scalar_t c, scalar_t d, scalar_t e, scalar_t f, scalar_t s, scalar_t t,
		scalar_t d_a, scalar_t d_b, scalar_t d_c, scalar_t d_d, scalar_t d_e, scalar_t d_f, scalar_t d_s, scalar_t d_tt){
  return d_a*s*s + 2.0*a*d_s*s + 
	 d_c*t*t + 2.0*c*d_tt*t + 
	 2.0*d_b*s*t + 2.0*b*d_s*t + 2*b*s*d_tt +
	 2.0*d_s*d + 2.0*s*d_d + 
	 2.0*d_e*t + 2.0*e*d_tt + d_f;  
}


/**
 * d_s/d_x
 */
 template <typename scalar_t>
__device__ scalar_t d_s_(scalar_t a, scalar_t b, scalar_t c, scalar_t d, scalar_t e, 
	scalar_t d_a, scalar_t d_b, scalar_t d_c, scalar_t d_d, scalar_t d_e,
       	scalar_t s_clamp, scalar_t t_clamp, scalar_t det){
  if (s_clamp==0) return 0;

  if (s_clamp+t_clamp<=1){
    scalar_t d_det = d_a*c + a*d_c - 2.0*b*d_b;
    scalar_t det2 = det*det;
    if (det2<eps) det2=eps;
    return ((d_b*e + b*d_e - d_c*d - c*d_d)*det -  (b*e-c*d)*d_det ) / ( det2 ); 
  }else if (s_clamp + t_clamp >1 && t_clamp > 0){
    scalar_t tmp = b*e - c*d + b*d - a*e;
    return ((d_b*e + b*d_e - d_c*d - c*d_d)*(b*d - a*e) - (b*e-c*d)*(d_b*d + b*d_d - d_a*e - a*d_e) ) / (tmp*tmp);
  }else{
    return 0;
  }
}


/**
 * d_t/d_x
 */
 template <typename scalar_t>
__device__ scalar_t d_t_(scalar_t a, scalar_t b, scalar_t c, scalar_t d, scalar_t e, 
	scalar_t d_a, scalar_t d_b, scalar_t d_c, scalar_t d_d, scalar_t d_e, 
       	scalar_t s_clamp, scalar_t t_clamp, scalar_t det){
  if (t_clamp==0) return 0;

  if (s_clamp+t_clamp<=1){
    scalar_t d_det = d_a*c + a*d_c - 2.0*b*d_b;
    scalar_t det2 = det*det;
    if (det2<eps) det2=eps;
    return ((d_b*d + b*d_d - d_a*e - a*d_e)*det -  (b*d-a*e)*d_det ) / ( det2 ); 
  }else if (s_clamp + t_clamp >1 && s_clamp > 0){
    scalar_t tmp = b*e - c*d + b*d - a*e;
    return ((d_b*d + b*d_d - d_a*e - a*d_e)*(b*e - c*d) - (b*d-a*e)*(d_b*e + b*d_e - d_c*d - c*d_d)) / (tmp * tmp);
  }else{
    return 0;
  }
}



/**
 * grad_triangle_to_offset
 */
 template <typename scalar_t>
__device__ void grad_triangle_to_offset(const scalar_t *grad_triangle, scalar_t *grad_offset, const int W, const int H, const int D, const int i, const int j, const int k, const int t, const scalar_t count){
  // for triangles in a single toplogy
  for (int tri_ind = 0; tri_ind<acceptTopology[1][t]; tri_ind++){
    // for vertices on the triangle
    for (int vertex_ind = 0; vertex_ind<3; vertex_ind++){

	// every vertex only contributes to the gradient of a single variable on the offset map
	int topology_ind = acceptTopology[0][t];
        int vertex = triTable[topology_ind][tri_ind*3+vertex_ind];

	atomicAdd( &grad_offset[vertices_to_offset[vertex][0]*W*H*D + 
			       (vertices_to_offset[vertex][1]+i)*H*D + 
  		               (vertices_to_offset[vertex][2]+j)*D +
  			        vertices_to_offset[vertex][3]+k], 
		   -grad_triangle[ tri_ind*9 + vertex_ind*3 + vertices_to_offset[vertex][0] ]/count );

	}
  }
}


/*
 * Compute the distance between a single point and a single triangle
 * params: 
 *  	triangle: 1D vector, length 3*3, 3 vertices in [0,1,2],[3,4,5],[6,7,8]
 *      point:    1D vector, length 3
 *      distance: scalar value
 */
 template <typename scalar_t>
__device__ scalar_t point_triangle_distance_forward(const scalar_t *triangle, const scalar_t *point)
{
 
  scalar_t det, s, t, sqrdistance;

  scalar_t B[3] = {triangle[0], triangle[1], triangle[2]}; 
  scalar_t E0[3] = {triangle[3]-B[0], triangle[4]-B[1], triangle[5]-B[2]};
  scalar_t E1[3] = {triangle[6]-B[0], triangle[7]-B[1], triangle[8]-B[2]};

  scalar_t a = E0[0]*E0[0] + E0[1]*E0[1] + E0[2]*E0[2];
  scalar_t b = E0[0]*E1[0] + E0[1]*E1[1] + E0[2]*E1[2];
  scalar_t c = E1[0]*E1[0] + E1[1]*E1[1] + E1[2]*E1[2];

  scalar_t D[3] = {B[0]-point[0], B[1]-point[1], B[2]-point[2]};
  scalar_t d = E0[0]*D[0] + E0[1]*D[1] + E0[2]*D[2];
  scalar_t e = E1[0]*D[0] + E1[1]*D[1] + E1[2]*D[2];
  scalar_t f = D[0]*D[0] + D[1]*D[1] + D[2]*D[2];

  det = a*c - b*b;
  if (det<eps) det=eps;
  s = (b*e - c*d) / det;
  t = (b*d - a*e) / det;

  if (s<0) s=0;
  if (t<0) t=0;
  scalar_t norm = s+t;
  if (norm>1){
          s = s/norm;
          t = t/norm;
  }

  sqrdistance = s * ( a*s + b*t + 2.0*d ) + t * ( b*s + c*t + 2.0*e ) + f;
  return sqrdistance;
}

/*
 * Backward function, compute the gradient on a single triangle w.r.t. the distance given a single point 
 * params: 
 * 	grad_output_ 	scalar_t, gradient on a point
 *  	triangle: 	1D vector, length 3*3, 3 vertices in [0,1,2],[3,4,5],[6,7,8]
 *      point:    	1D vector, length 3
 *  	grad_triangle: 	1D vector, length 3*3, 3 vertices in [0,1,2],[3,4,5],[6,7,8]
 *   	
 */
 template <typename scalar_t>
__device__ void point_triangle_distance_backward(const scalar_t grad_output_, const scalar_t *triangle, const scalar_t *point, scalar_t *grad_triangle)
{
  scalar_t det, s, t;

  scalar_t t11, t12, t13, t21, t22, t23, t31, t32, t33;
  scalar_t p1, p2, p3;
  t11 = triangle[0];
  t21 = triangle[1];
  t31 = triangle[2];
  t12 = triangle[3];
  t22 = triangle[4];
  t32 = triangle[5];
  t13 = triangle[6];
  t23 = triangle[7];
  t33 = triangle[8];

  p1 = point[0];
  p2 = point[1];
  p3 = point[2];

  scalar_t B[3] = {triangle[0], triangle[1], triangle[2]}; 
  scalar_t E0[3] = {triangle[3]-B[0], triangle[4]-B[1], triangle[5]-B[2]};
  scalar_t E1[3] = {triangle[6]-B[0], triangle[7]-B[1], triangle[8]-B[2]};

  scalar_t a = E0[0]*E0[0] + E0[1]*E0[1] + E0[2]*E0[2];
  scalar_t b = E0[0]*E1[0] + E0[1]*E1[1] + E0[2]*E1[2];
  scalar_t c = E1[0]*E1[0] + E1[1]*E1[1] + E1[2]*E1[2];


  scalar_t d_t11,d_t21,d_t31,d_t12,d_t22,d_t32,d_t13,d_t23,d_t33;
  d_t11=d_t21=d_t31=d_t12=d_t22=d_t32=d_t13=d_t23=d_t33 = 0;


  scalar_t D[3] = {B[0]-p1, B[1]-p2, B[2]-p3};
  scalar_t d = E0[0]*D[0] + E0[1]*D[1] + E0[2]*D[2];
  scalar_t e = E1[0]*D[0] + E1[1]*D[1] + E1[2]*D[2];
  scalar_t f = D[0]*D[0] + D[1]*D[1] + D[2]*D[2];

  det = a*c - b*b;
  if (det<eps) det=eps;
  s = (b*e - c*d) / det;
  t = (b*d - a*e) / det;

  scalar_t d_a,d_b,d_c,d_d,d_e,d_f;
  scalar_t s_clamp = s; 
  scalar_t t_clamp = t;
  if (s<0) s_clamp=0;
  if (t<0) t_clamp=0;
  scalar_t s_norm = s_clamp;
  scalar_t t_norm = t_clamp;
  scalar_t norm = s_clamp+t_clamp;
  if (norm>1){
          s_norm = s_clamp/norm;
          t_norm = t_clamp/norm;
  }

  // t11
  d_a = 2*t11 - 2*t12; d_b = 2*t11 - t12 - t13; d_c = 2*t11 - 2*t13; d_d = p1 - 2*t11 + t12; d_e = p1 - 2*t11 + t13; d_f = 2*t11 - 2*p1; 
  d_t11 += grad_output_ * d_sqrdistance_(a,b,c,d,e,f,s_norm,t_norm, d_a,d_b,d_c,d_d,d_e,d_f, d_s_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det), d_t_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det));
  // t21
  d_a = 2*t21 - 2*t22; d_b = 2*t21 - t22 - t23; d_c = 2*t21 - 2*t23; d_d = p2 - 2*t21 + t22; d_e = p2 - 2*t21 + t23; d_f = 2*t21 - 2*p2; 
  d_t21 += grad_output_ * d_sqrdistance_(a,b,c,d,e,f,s_norm,t_norm, d_a,d_b,d_c,d_d,d_e,d_f, d_s_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det), d_t_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det));
  // t31
  d_a = 2*t31 - 2*t32; d_b = 2*t31 - t32 - t33; d_c = 2*t31 - 2*t33; d_d = p3 - 2*t31 + t32; d_e = p3 - 2*t31 + t33; d_f = 2*t31 - 2*p3; 
  d_t31 += grad_output_ * d_sqrdistance_(a,b,c,d,e,f,s_norm,t_norm, d_a,d_b,d_c,d_d,d_e,d_f, d_s_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det), d_t_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det));

  // t12
  d_a = 2*t12 - 2*t11; d_b = t13 - t11; d_c = 0.0; d_d = t11 - p1; d_e = 0.0; d_f = 0.0; 
  d_t12 += grad_output_ * d_sqrdistance_(a,b,c,d,e,f,s_norm,t_norm, d_a,d_b,d_c,d_d,d_e,d_f, d_s_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det), d_t_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det));
  // t22
  d_a = 2*t22 - 2*t21; d_b = t23 - t21; d_c = 0.0; d_d = t21 - p2; d_e = 0.0; d_f = 0.0; 
  d_t22 += grad_output_ * d_sqrdistance_(a,b,c,d,e,f,s_norm,t_norm, d_a,d_b,d_c,d_d,d_e,d_f, d_s_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det), d_t_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det));
  // t32
  d_a = 2*t32 - 2*t31; d_b = t33 - t31; d_c = 0.0; d_d = t31 - p3; d_e = 0.0; d_f = 0.0; 
  d_t32 += grad_output_ * d_sqrdistance_(a,b,c,d,e,f,s_norm,t_norm, d_a,d_b,d_c,d_d,d_e,d_f, d_s_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det), d_t_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det));

  // t13
  d_a = 0.0; d_b = t12 - t11; d_c = 2*t13 - 2*t11; d_d = 0.0; d_e = t11 - p1; d_f = 0.0; 
  d_t13 += grad_output_ * d_sqrdistance_(a,b,c,d,e,f,s_norm,t_norm, d_a,d_b,d_c,d_d,d_e,d_f, d_s_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det), d_t_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det));
  // t23
  d_a = 0.0; d_b = t22 - t21; d_c = 2*t23 - 2*t21; d_d = 0.0; d_e = t21 - p2; d_f = 0.0; 
  d_t23 += grad_output_ * d_sqrdistance_(a,b,c,d,e,f,s_norm,t_norm, d_a,d_b,d_c,d_d,d_e,d_f, d_s_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det), d_t_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det));
  // t33
  d_a = 0.0; d_b = t32 - t31; d_c = 2*t33 - 2*t31; d_d = 0.0; d_e = t31 - p3; d_f = 0.0; 
  d_t33 += grad_output_ * d_sqrdistance_(a,b,c,d,e,f,s_norm,t_norm, d_a,d_b,d_c,d_d,d_e,d_f, d_s_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det), d_t_(a,b,c,d,e, d_a,d_b,d_c,d_d,d_e, s_clamp,t_clamp,det));

  grad_triangle[0] = d_t11;
  grad_triangle[1] = d_t21;
  grad_triangle[2] = d_t31;

  grad_triangle[3] = d_t12;
  grad_triangle[4] = d_t22;
  grad_triangle[5] = d_t32;

  grad_triangle[6] = d_t13;
  grad_triangle[7] = d_t23;
  grad_triangle[8] = d_t33;

}


/* 
 * cuda kernel, parallel over per cell per topology
 */
 template <typename scalar_t>
__global__ void point_toplogy_distance_kernel(const scalar_t *offset, const scalar_t *points, scalar_t *distances, int *indices, const int n){
  // topology
  int t = threadIdx.x;
  int topology_ind = acceptTopology[0][t];
  int T = blockDim.x + 1;
  // cell indices
  int i = blockIdx.x;
  int j = blockIdx.y;
  int k = blockIdx.z;
  // cell size
  int Wc = gridDim.x;
  int Hc = gridDim.y;
  int Dc = gridDim.z;
  int ind = i*Hc*Dc + j*Dc + k;
  // offset size, note that we always have 3x(W+1)x(H+1)x(Dx1) offset for WxHxD grid
  int W = Wc + 1;
  int H = Hc + 1;
  int D = Dc + 1;

  // offset_to_vertices
  scalar_t vertices[12*3];
  offset_to_vertices_cuda(offset, W, H, D, i, j, k, vertices);

  //scalar_t *triangle = offset_to_triangles(offset, i, j, k, t);

  scalar_t distance_sum=0.0;
  scalar_t count=0;

  for (int p=0; p<n; p++){
     scalar_t px = points[p*3+0];
     scalar_t py = points[p*3+1];
     scalar_t pz = points[p*3+2];
     // if point is inside of the grid
     if (px >= i && px < i+grid_size && py >= j && py < j+grid_size && pz >= k && pz < k+grid_size){
       
       // min distance to a triangle in the same topology
       // also save the min indice for back-propagation
       scalar_t min_distance = 10000.0;
       long int min_indice = -1;

       for (int tri_ind = 0; tri_ind<acceptTopology[1][t]; tri_ind++){
         // offset_to_triangles
	 // Note: offset_to_triangles is inside of the loop to avoid dynamically allocate memory, different to cpu version
	 scalar_t triangle_single[3*3] = { // v1 
		 			vertices[triTable[topology_ind][tri_ind*3+0]*3 + 0] + scalar_t(i),
				        vertices[triTable[topology_ind][tri_ind*3+0]*3 + 1] + scalar_t(j),
				        vertices[triTable[topology_ind][tri_ind*3+0]*3 + 2] + scalar_t(k), 
					// v2
		 			vertices[triTable[topology_ind][tri_ind*3+1]*3 + 0] + scalar_t(i),
				        vertices[triTable[topology_ind][tri_ind*3+1]*3 + 1] + scalar_t(j),
				        vertices[triTable[topology_ind][tri_ind*3+1]*3 + 2] + scalar_t(k), 
					// v3
		 			vertices[triTable[topology_ind][tri_ind*3+2]*3 + 0] + scalar_t(i),
				        vertices[triTable[topology_ind][tri_ind*3+2]*3 + 1] + scalar_t(j),
				        vertices[triTable[topology_ind][tri_ind*3+2]*3 + 2] + scalar_t(k) }; 
	 scalar_t point_single[3] = {px, py, pz};
         scalar_t distance_single = point_triangle_distance_forward( triangle_single, point_single );

	 if (distance_single < min_distance){
           min_distance = distance_single;
	   min_indice = tri_ind;
	 }

       }
       indices[p*T + t] = min_indice;
       distance_sum += min_distance;
       count += 1;
     }
  }
  // if the current grid is not empty
  if (count>0) {
    distances[ind*T + t] = distance_sum/count;
  } else {
    distances[ind*T + t] = distance_empty; 
  }

  __syncthreads();

}



/* 
 * cuda kernel, parallel over per cell per topology
 */
 template <typename scalar_t>
__global__ void grad_point_toplogy_distance_kernel(const scalar_t *grad_output, const scalar_t *offset, const scalar_t *points, const int *indices, scalar_t *grad_offset, const int n){

  // topology
  int t = threadIdx.x;
  int T = blockDim.x + 1;
  int topology_ind = acceptTopology[0][t];
  // cell indices
  int i = blockIdx.x;
  int j = blockIdx.y;
  int k = blockIdx.z;
  // cell size
  int Wc = gridDim.x;
  int Hc = gridDim.y;
  int Dc = gridDim.z;
  int ind = i*Hc*Dc + j*Dc + k;

  int grad_ind = ind*T + t; 
  const scalar_t grad_output_element = grad_output[grad_ind];
  //printf("%d %d %d, %d, grad_output_element %f\n", i, j, k, grad_ind, grad_output_element );
  // offset size, note that we always have 3x(W+1)x(H+1)x(Dx1) offset for WxHxD grid
  int W = Wc + 1;
  int H = Hc + 1;
  int D = Dc + 1;


  // offset_to_vertices
  scalar_t vertices[12*3];
  offset_to_vertices_cuda(offset, W, H, D, i, j, k, vertices);

  //scalar_t *triangle = offset_to_triangles(offset, i, j, k, t);

  scalar_t count=0;

  // allocate memory for accumulating the gradients
  // assuming maximum number of triangles for each topology is 4 as in Marching Cubes
  scalar_t grad_triangle_all[4*3*3] = {0};


  for (int p=0; p<n; p++){
     scalar_t px = points[p*3+0];
     scalar_t py = points[p*3+1];
     scalar_t pz = points[p*3+2];
     // if point is inside of the grid
     if (px >= scalar_t(i) && px < scalar_t(i)+grid_size && py >= scalar_t(j) && py < scalar_t(j)+grid_size && pz >= scalar_t(k) && pz < scalar_t(k)+grid_size){

       // printf("(%f %f %f) in [%f %f %f]\n", px, py, pz, scalar_t(i), scalar_t(j), scalar_t(k) );
       // printf("grad_output_element %f\n", grad_output_element );
       // printf("grad_output_element index: %d*%d + %d = %d\n", ind, T, t, ind*T+t);
       // only back propagate to the nearest triangle
       int tri_ind = indices[p*T + t];
       if (tri_ind == -1) continue;
       // offset_to_triangles
       // Note: offset_to_triangles is inside of the loop to avoid dynamically allocate memory, different to cpu version
       scalar_t triangle_single[3*3] = { // v1 
           			vertices[triTable[topology_ind][tri_ind*3+0]*3 + 0] + scalar_t(i),
          		        vertices[triTable[topology_ind][tri_ind*3+0]*3 + 1] + scalar_t(j),
          		        vertices[triTable[topology_ind][tri_ind*3+0]*3 + 2] + scalar_t(k), 
          			// v2
           			vertices[triTable[topology_ind][tri_ind*3+1]*3 + 0] + scalar_t(i),
          		        vertices[triTable[topology_ind][tri_ind*3+1]*3 + 1] + scalar_t(j),
          		        vertices[triTable[topology_ind][tri_ind*3+1]*3 + 2] + scalar_t(k), 
          			// v3
           			vertices[triTable[topology_ind][tri_ind*3+2]*3 + 0] + scalar_t(i),
          		        vertices[triTable[topology_ind][tri_ind*3+2]*3 + 1] + scalar_t(j),
          		        vertices[triTable[topology_ind][tri_ind*3+2]*3 + 2] + scalar_t(k) }; 
        scalar_t point_single[3] = {px, py, pz};
	scalar_t grad_triangle[3*3];
        //point_triangle_distance_backward(grad_output[ind*T + t], triangle_single, point_single, grad_triangle);
        point_triangle_distance_backward(grad_output_element, triangle_single, point_single, grad_triangle);

	// accumulate gradients over all the points for each triangle
        // to reduce times of updating global memory	
	for (int gi=0; gi<9; gi++){
		grad_triangle_all[tri_ind*9 + gi] += grad_triangle[gi];
	}

	count += 1.0;
      }
  }

  if (count<1.0) return;

  // 
  grad_triangle_to_offset(grad_triangle_all, grad_offset, W, H, D, i, j, k, t, count);

}

/* 
 * check all distances and assign a large loss to the empty topology if the cell is not empty
 * params:
 * 	  distances 	point to line-segment distance
 * 	  T 		number of all acceptable topologies	
 */
 template <typename scalar_t>
__global__ void update_empty_topology(scalar_t *distances, const int T){
  // cell indices
  int i = blockIdx.x;
  int j = blockIdx.y;
  int k = blockIdx.z;
  // cell size
  int Hc = gridDim.y;
  int Dc = gridDim.z;
  int ind = i*Hc*Dc + j*Dc + k;

  int empty=1;
  scalar_t max_distance = -1.0;
  for (int t=0; t<T-1; t++){
    scalar_t d = distances[ind*T + t];
    if (d!=distance_empty) empty = 0;
    if (d>max_distance) max_distance = d; 
  }

  if (empty==0) {
	  distances[ind*T + T-1] = max_distance*10.0;
  }

  __syncthreads();

}  


} //namespace


/*
 * Forward function, calculating the point to mesh distances for all grids
 * params: 
 * 	  offset 	input, offset map for x,y,z directions, 3x(W+1)x(H+1)x(D+1) 
 *  	points 	input, all points, N_allx3
 *  	distances  	output, point to mesh distances for every grid for every topolopy, (WxHxD)xT 
 *  	indices_all   output, to record which triangle in each topology is the nearest one for backpropagation, N_allxT
 */	
void point_topology_distance_kernel_forward( 
      at::Tensor offset,
      at::Tensor points,
      at::Tensor distances,
      at::Tensor indices_all){
  int W = offset.size(1)-1;
  int H = offset.size(2)-1;
  int D = offset.size(3)-1;
  int T = distances.size(1);
  dim3 dimGrid(W, H, D);
  dim3 dimBlock(T-1, 1, 1);
  int n = points.size(0);
  assert(offset.type().scalarType() == at::ScalarType::Float);
  assert(points.type().scalarType() == at::ScalarType::Float);
  assert(distances.type().scalarType() == at::ScalarType::Float);
  assert(indices_all.type().scalarType() == at::ScalarType::Int);
  // lauch the kernel
  point_toplogy_distance_kernel<float><<< dimGrid, dimBlock>>>(
      offset.data<float>(),
      points.data<float>(),
      distances.data<float>(),
      indices_all.data<int>(),
      n);
  
  update_empty_topology<float><<<dimGrid, 1>>>(
      distances.data<float>(),
      T);
}

/*
 * Backward function, calculating the gradients for the full offset map 
 * params: 
 *  	  grad_output   input, gradient on the output distances, (WxHxD)xT
 * 	    offset      	input, offset map for x,y,z directions, 3x(W+1)x(H+1)x(D+1) 
 *  	  points       	input, all points, N_allx3
 *  	  indices_all   input, recorded which triangle in each topology is the nearest one for backpropagation, N_allxT
 *  	  grad_offset  	output, gradient on the full offset map, 3x(W+1)x(H+1)x(D+1)  
 *
 */	
void point_topology_distance_kernel_backward( 
      at::Tensor grad_output,
      at::Tensor offset,
      at::Tensor points,
      at::Tensor indices_all,
      at::Tensor grad_offset){
  int W = offset.size(1) - 1;
  int H = offset.size(2) - 1;
  int D = offset.size(3) - 1;
  int T = grad_output.size(1);
  dim3 dimGrid(W, H, D);
  dim3 dimBlock(T-1, 1, 1);
  int n = points.size(0);
  
  assert(offset.type().scalarType() == at::ScalarType::Float);
  assert(points.type().scalarType() == at::ScalarType::Float);
  assert(grad_output.type().scalarType() == at::ScalarType::Float);
  assert(indices_all.type().scalarType() == at::ScalarType::Int);
  assert(grad_offset.type().scalarType() == at::ScalarType::Float);
  // lauch the kernel
  grad_point_toplogy_distance_kernel<float><<<dimGrid, dimBlock>>>(
      grad_output.data<float>(),
      offset.data<float>(),
      points.data<float>(),
      indices_all.data<int>(),
      grad_offset.data<float>(),
      n);
}
