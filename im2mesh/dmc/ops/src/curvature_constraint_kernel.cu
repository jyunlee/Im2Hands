#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#include <typeinfo>

// number of all triangles for topologies up to __3__ triangles
#define NumTri 220 
#define NumTop 96 

__constant__ float eps=1e-6;

__constant__ float thres=1e-4;

// up to __3__ triangles
__constant__ int acceptTopologyWithFlip[2][96]={ {1, 2, 3, 4, 6, 7, 8, 9, 11, 12, 13, 14, 15, 16, 17, 19, 25, 31, 32, 34, 35, 38, 47, 48, 49, 50, 51, 55, 59, 63, 64, 68, 70, 76, 79, 96, 98, 100, 102, 103, 110, 111, 112, 115, 118, 119, 127, 0, 255, 128, 136, 137, 140, 143, 144, 145, 152, 153, 155, 157, 159, 176, 179, 185, 187, 191, 192, 196, 200, 204, 205, 206, 207, 208, 217, 220, 221, 223, 224, 230, 236, 238, 239, 240, 241, 242, 243, 244, 246, 247, 248, 249, 251, 252, 253, 254},
				    {1, 1, 2, 1, 2, 3, 1, 2, 3, 2, 3, 3, 2, 1, 2, 3, 3, 3, 1, 2, 3, 3, 3, 2, 3, 3, 2, 3, 3, 2, 1, 2, 3, 3, 3, 2, 3, 3, 2, 3, 3, 2, 3, 3, 3, 2, 1, 0, 0, 1, 2, 3, 3, 3, 2, 3, 3, 2, 3, 3, 2, 3, 3, 3, 2, 1, 2, 3, 3, 2, 3, 3, 2, 3, 3, 3, 2, 1, 3, 3, 3, 2, 1, 2, 3, 3, 2, 3, 2, 1, 3, 2, 1, 2, 1, 1}};

// look-up-table in Marching Cubes Algorithm
__constant__ int triTable[256][16] =
{{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1},
{3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1},
{3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1},
{3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1},
{9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1},
{9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
{2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1},
{8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1},
{9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
{4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1},
{3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1},
{1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1},
{4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1},
{4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
{5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1},
{2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1},
{9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
{0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
{2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1},
{10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1},
{5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1},
{5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1},
{9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1},
{0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1},
{1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1},
{10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1},
{8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1},
{2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1},
{7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1},
{2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1},
{11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1},
{5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1},
{11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1},
{11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
{1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1},
{9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1},
{5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1},
{2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
{5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1},
{6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1},
{3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1},
{6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1},
{5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1},
{1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
{10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1},
{6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1},
{8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1},
{7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1},
{3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
{5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1},
{0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1},
{9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1},
{8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1},
{5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1},
{0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1},
{6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1},
{10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1},
{10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1},
{8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1},
{1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1},
{0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1},
{10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1},
{3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1},
{6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1},
{9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1},
{8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1},
{3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1},
{6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1},
{0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1},
{10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1},
{10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1},
{2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1},
{7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1},
{7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1},
{2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1},
{1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1},
{11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1},
{8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1},
{0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1},
{7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
{10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
{2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
{6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1},
{7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1},
{2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1},
{1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1},
{10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1},
{10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1},
{0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1},
{7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1},
{6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1},
{8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1},
{9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1},
{6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1},
{4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1},
{10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1},
{8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1},
{0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1},
{1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1},
{8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1},
{10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1},
{4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1},
{10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
{5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
{11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1},
{9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
{6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1},
{7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1},
{3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1},
{7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1},
{3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1},
{6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1},
{9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1},
{1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1},
{4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1},
{7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1},
{6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1},
{3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1},
{0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1},
{6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1},
{0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1},
{11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1},
{6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1},
{5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1},
{9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1},
{1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1},
{1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1},
{10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1},
{0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1},
{5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1},
{10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1},
{11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1},
{9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1},
{7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1},
{2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1},
{8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1},
{9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1},
{9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1},
{1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1},
{9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1},
{9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1},
{5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1},
{0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1},
{10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1},
{2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1},
{0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1},
{0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1},
{9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1},
{5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1},
{3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1},
{5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1},
{8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1},
{0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1},
{9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1},
{1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1},
{3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1},
{4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1},
{9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1},
{11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1},
{11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1},
{2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1},
{9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1},
{3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1},
{1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1},
{4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1},
{3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1},
{0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1},
{9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1},
{1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1}};

__constant__ int vertices_to_offset[12][4]={ {0, 1, 1, 0}, // #0
				{1, 1, 1, 0}, // #1
				{0, 1, 0, 0}, // #2
				{1, 0, 1, 0}, // #3
				
				{0, 1, 1, 1}, // #4
				{1, 1, 1, 1}, // #5
				{0, 1, 0, 1}, // #6
				{1, 0, 1, 1}, // #7

				{2, 0, 1, 1}, // #8
				{2, 1, 1, 1}, // #9
				{2, 1, 0, 1}, // #10
				{2, 0, 0, 1}}; // #11

namespace{
/**
 * get the vertex locations from the vertex displacement field
 */
 template <typename scalar_t>
__device__ void offset_to_vertices_cuda(const scalar_t *offset, const int W, const int H, const int D, const int x, const int y, const int z, scalar_t *vertices){
  // #0
  vertices[0 ] = 0.5-offset[0       + (x+1)*H*D + (y+1)*D + z   ]; 
  vertices[1 ] = 1.0; 
  vertices[2 ] = 0.0; 
  // #1
  vertices[3 ] = 1.0; 
  vertices[4 ] = 0.5-offset[1*W*H*D + (x+1)*H*D + (y+1)*D + z   ]; 
  vertices[5 ] = 0.0; 
  // #2
  vertices[6 ] = 0.5-offset[0       + (x+1)*H*D + (y  )*D + z   ]; 
  vertices[7 ] = 0.0; 
  vertices[8 ] = 0.0; 
  // #3
  vertices[9 ] = 0.0; 
  vertices[10] = 0.5-offset[1*W*H*D + (x  )*H*D + (y+1)*D + z   ]; 
  vertices[11] = 0.0; 

  // #4
  vertices[12] = 0.5-offset[0       + (x+1)*H*D + (y+1)*D + z+1 ]; 
  vertices[13] = 1.0; 
  vertices[14] = 1.0; 
  // #5
  vertices[15] = 1.0; 
  vertices[16] = 0.5-offset[1*W*H*D + (x+1)*H*D + (y+1)*D + z+1 ]; 
  vertices[17] = 1.0; 
  // #6
  vertices[18] = 0.5-offset[0       + (x+1)*H*D + (y  )*D + z+1 ]; 
  vertices[19] = 0.0; 
  vertices[20] = 1.0; 
  // #7
  vertices[21] = 0.0; 
  vertices[22] = 0.5-offset[1*W*H*D + (x  )*H*D + (y+1)*D + z+1 ]; 
  vertices[23] = 1.0; 

  // #8
  vertices[24] = 0.0; 
  vertices[25] = 1.0; 
  vertices[26] = 0.5-offset[2*W*H*D + (x  )*H*D + (y+1)*D + z+1 ]; 
  // #9
  vertices[27] = 1.0; 
  vertices[28] = 1.0; 
  vertices[29] = 0.5-offset[2*W*H*D + (x+1)*H*D + (y+1)*D + z+1 ]; 
  // #10
  vertices[30] = 1.0; 
  vertices[31] = 0.0; 
  vertices[32] = 0.5-offset[2*W*H*D + (x+1)*H*D + (y  )*D + z+1 ]; 
  // #11
  vertices[33] = 0.0; 
  vertices[34] = 0.0; 
  vertices[35] = 0.5-offset[2*W*H*D + (x  )*H*D + (y  )*D + z+1 ]; 
}



/**
 * check the intersection between two integer lists 
 * param:
 * 	array1 		input, integer list denoting the vertex indices on a single face, length 4
 * 	array2 		input, integer list denoting the vertex indices of a triangle, length 3
 * 	out		output, intersected vertex indices, padded with -1 to a fixed length, length 3
 */
 //template <typename scalar_t>
__device__ void intersection(const int *array1, const int *array2, int *out){

    int count = 0;

    // initialization
    for (int i=0; i<3; i++){
        out[i] = -1;
    }
    

    for (int i=0; i<4; i++){
	for (int j=0; j<3; j++){
	    if (array2[j]==array1[i]){
		out[count] = array1[i]; 
		count ++;
	    }
	}
    }
}

/**
 * return the vertex indices on a given surface of a cell
 */
 //template <typename scalar_t>
__device__ void get_vertices_on_face(const int r, int *row){
    int vertices_on_location[6][4] = { {5, 9, 1, 10},
                                   {7, 8, 3, 11},
                                   {4, 9, 0, 8},
                                   {6, 10, 2, 11},
                                   {4, 5, 6, 7},
                                   {0, 1, 2, 3} };
    for (int i=0; i<4; i++){
      row[i]=vertices_on_location[r][i];
    }
}

/**
 * calculate dn/dpb
 * 	 0  		c3-a3  		-(c2-a2)
 * 	 -(c3-a3)  	0 		c1-a1
 * 	 c2-a2 		-(c1-a1)	0
 */
 template <typename scalar_t>
__device__ void dn_dpb( const scalar_t *vertices, const int a, const int c, const scalar_t *dn, scalar_t *db ){

  scalar_t d3 = vertices[c*3+2] - vertices[a*3+2];
  scalar_t d2 = vertices[c*3+1] - vertices[a*3+1];
  scalar_t d1 = vertices[c*3+0] - vertices[a*3+0];

  db[0] =             d3*dn[1] - d2*dn[2];
  db[1] = -d3*dn[0]            + d1*dn[2];
  db[2] =  d2*dn[0] - d1*dn[1]           ;

}

/**
 * dn/dpc
 *  	0  		-(b3-a3)  	b2-a2
 *  	(b3-a3)  	0 		-(b1-a1)
 *  	-(b2-a2) 	b1-a1		0
 */
 template <typename scalar_t>
__device__ void dn_dpc( const scalar_t *vertices, const int a, const int b, const scalar_t *dn, scalar_t *dc ){

  scalar_t d3 = vertices[b*3+2] - vertices[a*3+2];
  scalar_t d2 = vertices[b*3+1] - vertices[a*3+1];
  scalar_t d1 = vertices[b*3+0] - vertices[a*3+0];

  dc[0] =           - d3*dn[1] + d2*dn[2];
  dc[1] =  d3*dn[0]            - d1*dn[2];
  dc[2] = -d2*dn[0] + d1*dn[1]           ;

}

/**
 * calculate dn/dpa
 *  	0  		b3-c3  		-(b2-c2)
 *  	-(b3-c3)  	0 		b1-c1	
 *  	b2-c2 		-(b1-c1)	0
 */
 template <typename scalar_t>
__device__ void dn_dpa( const scalar_t *vertices, const int b, const int c, const scalar_t *dn, scalar_t *da ){

  scalar_t d3 = vertices[b*3+2] - vertices[c*3+2];
  scalar_t d2 = vertices[b*3+1] - vertices[c*3+1];
  scalar_t d1 = vertices[b*3+0] - vertices[c*3+0];

  da[0] =             d3*dn[1] - d2*dn[2];
  da[1] = -d3*dn[0]            + d1*dn[2];
  da[2] =  d2*dn[0] - d1*dn[1]           ;
}

/** 
 * offset_to_normals, return normal vectors of all triangles (NOT topologies)
 * params:
 * 	offset 		input
 * 	W 		input, number of cells on one of the directions
 * 	H 		input, number of cells on one of the directions
 * 	D 		input, number of cells on one of the directions
 *	i_ 		input, index of the cell on one of the directions
 *	j_ 		input, index of the cell on one of the directions
 *	k_ 		input, index of the cell on one of the directions
 *	location	input, indicating the relative location of the current cell in the pairwise loss
 * 				0: x1 
 * 				1: x2 
 * 				2: y1 
 * 				3: y2 
 * 				4: z1 
 * 				5: z2 
 * 				6: dummy case for inner cell loss 
 * 				7: dummy case for inner cell loss 
 * 	normal 		output
 * 	length 		output, return the length of the normal vector for computing the gradient
 */
 template <typename scalar_t>
__device__ void offset_to_normals(const scalar_t *offset, const int W, const int H, const int D, const int i_, const int j_, const int k_, const int location, scalar_t *normal, scalar_t *length){

  // offset_to_vertices
  scalar_t vertices[12*3];
  offset_to_vertices_cuda(offset, W, H, D, i_, j_, k_, vertices);


  int vertices_on_face[4];
  get_vertices_on_face(location, vertices_on_face); 

  int tri_cnt = 0;
  for (int i = 0; i < NumTop; i++){
      int top_ind = acceptTopologyWithFlip[0][i]; 
      int num_triangle = acceptTopologyWithFlip[1][i];
      for (int tri_ind = 0; tri_ind<num_triangle; tri_ind++){
	  // get the indices of the triangle vertices
	  int triangle[3] = {triTable[top_ind][tri_ind*3], triTable[top_ind][tri_ind*3+1], triTable[top_ind][tri_ind*3+2]};
	  
	  // check if the triangle has a line on the face we care about
	  // simply assign a dummy normal vector if not
          int inter_ind[3];
	  intersection(vertices_on_face, triangle, inter_ind);


	  // location > 5 means inner case instead of x, y, z direction
	  if (location>5 ||  (location <=5 && inter_ind[0]>-1 && inter_ind[1]>-1 && inter_ind[2]==-1) ){
	
	    // consider inside/outside, then the direction of the normal vector 
	    // decided by the look-up-table
	    int a, b, c;
	    a = triangle[0];
	    b = triangle[1];
	    c = triangle[2];
		
	    // compute the normal
	    scalar_t vec1[3] = { vertices[b*3+0] - vertices[a*3+0], 
		    	      vertices[b*3+1] - vertices[a*3+1],
		    	      vertices[b*3+2] - vertices[a*3+2] };
      scalar_t vec2[3] = { vertices[c*3+0] - vertices[a*3+0], 
		    	      vertices[c*3+1] - vertices[a*3+1],
		    	      vertices[c*3+2] - vertices[a*3+2] };

	    // cross product 
	    scalar_t cross[3] = { vec1[1]*vec2[2] - vec1[2]*vec2[1],
	    		       vec1[2]*vec2[0] - vec1[0]*vec2[2],
	    		       vec1[0]*vec2[1] - vec1[1]*vec2[0] };

	    // normalized to unit vector
	    scalar_t l2 = sqrt(cross[0]*cross[0] + cross[1]*cross[1] + cross[2]*cross[2]);
	    if (l2<eps) { l2=eps; }
	    
	    // copy to the normal vector, which saved the normal of all triangles
	    normal[tri_cnt*3 + 0] = cross[0]/l2;
	    normal[tri_cnt*3 + 1] = cross[1]/l2;
	    normal[tri_cnt*3 + 2] = cross[2]/l2;

	    length[tri_cnt] = l2;

	  }
	  else{
            // set dummy normal vector
	    normal[tri_cnt*3 + 0] = 1.0;
	    normal[tri_cnt*3 + 1] = 1.0;
	    normal[tri_cnt*3 + 2] = 1.0;
	  }

	  tri_cnt ++;
      }
  }
}

/**
 * calculate the gradient back-propagated to the offset
 */
 template <typename scalar_t>
__device__ void grad_normal_to_offset(scalar_t *grad_offset, const scalar_t *grad_normal, const scalar_t *offset, const int W, const int H, const int D, const int i_, const int j_, int k_,  const int location){

  // offset_to_vertices
  scalar_t vertices[12*3];
  offset_to_vertices_cuda(offset, W, H, D, i_, j_, k_, vertices);


  int vertices_on_face[4];
  get_vertices_on_face(location, vertices_on_face); 


  int tri_cnt = 0;
  for (int i = 0; i < NumTop; i++){
      int top_ind = acceptTopologyWithFlip[0][i]; 
      int num_triangle = acceptTopologyWithFlip[1][i];
      for (int tri_ind = 0; tri_ind<num_triangle; tri_ind++){

	  // get the gradient on the normal vector of the current triangle
	  scalar_t grad_tri[3] = {grad_normal[tri_cnt*3 + 0], grad_normal[tri_cnt*3 +1], grad_normal[tri_cnt*3 + 2]};

	  // get the indices of the triangle vertices
	  int triangle[3] = {triTable[top_ind][tri_ind*3], triTable[top_ind][tri_ind*3+1], triTable[top_ind][tri_ind*3+2]};
	  
	  // check if the triangle has a line on the face we care about
	  // simply assign a dummy normal vector if not
          int inter_ind[3];
	  intersection(vertices_on_face, triangle, inter_ind);


	  // location > 5 means inner case instead of x, y, z direction
	  if (location>5 ||  (location <=5 && inter_ind[0]>-1 && inter_ind[1]>-1 && inter_ind[2]==-1) ){

	    // consider inside/outside, then the direction of the normal vector 
	    // decided by the look-up-table
	    int a, b, c;
	    a = triangle[0];
	    b = triangle[1];
	    c = triangle[2];

	    // dn_da
	    scalar_t da[3];
  	    dn_dpa(vertices, b, c, grad_tri, da);

	    atomicAdd( &grad_offset[ vertices_to_offset[a][0]*W*H*D + 
	    		            (vertices_to_offset[a][1]+i_)*H*D + 
  	    	                    (vertices_to_offset[a][2]+j_)*D +
  	    		             vertices_to_offset[a][3]+k_], 
		       da[vertices_to_offset[a][0]] );

	    // dn_db
	    scalar_t db[3];
  	    dn_dpb(vertices, a, c, grad_tri, db);

	    atomicAdd( &grad_offset[ vertices_to_offset[b][0]*W*H*D + 
	    		            (vertices_to_offset[b][1]+i_)*H*D + 
  	    	                    (vertices_to_offset[b][2]+j_)*D +
  	    		             vertices_to_offset[b][3]+k_], 
		       db[vertices_to_offset[b][0]] );

	    // dn_dc
	    scalar_t dc[3];
  	    dn_dpc(vertices, a, b, grad_tri, dc);

	    atomicAdd( &grad_offset[ vertices_to_offset[c][0]*W*H*D + 
	    		            (vertices_to_offset[c][1]+i_)*H*D + 
  	    	                    (vertices_to_offset[c][2]+j_)*D +
  	    		             vertices_to_offset[c][3]+k_], 
		       dc[vertices_to_offset[c][0]] );

	  }

	  tri_cnt++;
      }
  }
}

/**
 * calculate d(normalized normal vector)/d(normal vector)
 */
 template <typename scalar_t>
__device__ void grad_normalized_to_normal(scalar_t *grad_normal, const scalar_t *normal, const scalar_t l){

  scalar_t orig_normal[3] = {normal[0]*l, normal[1]*l, normal[2]*l};
  
  scalar_t l3 = l*l*l;
  scalar_t g00 = (orig_normal[1]*orig_normal[1] + orig_normal[2]*orig_normal[2])/l3;
  scalar_t g01 = -orig_normal[0]*orig_normal[1]/l3; 
  scalar_t g02 = -orig_normal[0]*orig_normal[2]/l3; 
  scalar_t g10 = g01; 
  scalar_t g11 = (orig_normal[0]*orig_normal[0] + orig_normal[2]*orig_normal[2])/l3;
  scalar_t g12 = -orig_normal[1]*orig_normal[2]/l3; 
  scalar_t g20 = g02;
  scalar_t g21 = g12;
  scalar_t g22 = (orig_normal[0]*orig_normal[0] + orig_normal[1]*orig_normal[1])/l3;

  scalar_t g1 = grad_normal[0]*g00 + grad_normal[1]*g01 + grad_normal[2]*g02;
  scalar_t g2 = grad_normal[0]*g10 + grad_normal[1]*g11 + grad_normal[2]*g12;
  scalar_t g3 = grad_normal[0]*g20 + grad_normal[1]*g21 + grad_normal[2]*g22;

  grad_normal[0] = g1;
  grad_normal[1] = g2;
  grad_normal[2] = g3;
}

/**
 * calculate the loss between two neighboring cells
 * params:
 * 	offset 		input, the vertex displacement field of the full grid
 * 	topolopy 	input, probability for each triangle'
 * 	mask 		input, mask denoting if two topogolies have connected triangles or not	
 * 	loss 		output, curvature loss
 * 	direction	input, a integer denoting the neighoring relationship between two cells 
 * 				0: two cells adajecent in x direction
 * 				1: two cells adajecent in y direction
 * 				2: two cells adajecent in z direction
 * 				3: dummy label for inner cell loss
 */
 template <typename scalar_t>
__global__ void pairwise_loss(const scalar_t *offset, const scalar_t *topology, const scalar_t *mask, scalar_t *loss, const int direction){
  int i1 = blockIdx.x;
  int j1 = blockIdx.y;
  int k1 = threadIdx.x;

  int W = gridDim.x;
  int H = gridDim.y;
  int D = blockDim.x;

  int T = NumTri;
  int i2=0, j2=0, k2=0, ind1=0, ind2=0;
  // x direction
  if (direction==0){
       if (i1==W-1) return;
       ind1 = i1*H*D + j1*H + k1;
       ind2 = ind1+H*D;
       i2 = i1+1;
       j2 = j1;
       k2 = k1;
  }
  // y direction
  else if (direction==1){
       if (j1==H-1) return;
       ind1 = i1*H*D + j1*H + k1;
       ind2 = ind1+H;
       i2 = i1;
       j2 = j1+1;
       k2 = k1;
  }
  // z direction
  else if (direction==2){
       if (k1==D-1) return;
       ind1 = i1*H*D + j1*H + k1;
       ind2 = ind1+1;
       i2 = i1;
       j2 = j1;
       k2 = k1+1;
  }
  // inner loss, within the same cell
  else if (direction==3){
       ind1 = i1*H*D + j1*H + k1;
       ind2 = ind1;
       i2 = i1;
       j2 = j1;
       k2 = k1;
  }
  // get normal vector in both grids
  scalar_t norm1[NumTri*3];
  scalar_t norm2[NumTri*3];
  scalar_t length1[NumTri];
  scalar_t length2[NumTri];
  offset_to_normals(offset, W+1, H+1, D+1, i1, j1, k1, direction*2, norm1, length1);
  offset_to_normals(offset, W+1, H+1, D+1, i2, j2, k2, direction*2+1, norm2, length2);


  scalar_t loss_=0;


  for (int ti=0; ti<T; ti++){

    for (int tj=0; tj<T; tj++){

       // no loss if two topologies are not connected
       scalar_t conn_ij = mask[ti*T + tj];
       if (conn_ij<eps) continue;

       // joint probability of two topology combinations
       // corresponding to outer product
       scalar_t pi = topology[ind1*T+ti];
       scalar_t pj = topology[ind2*T+tj];
       scalar_t p_ij = pi*pj;

       // l2 loss
       scalar_t diff_norm0 = norm1[ti*3 + 0]-norm2[tj*3 + 0];
       scalar_t diff_norm1 = norm1[ti*3 + 1]-norm2[tj*3 + 1];
       scalar_t diff_norm2 = norm1[ti*3 + 2]-norm2[tj*3 + 2];

       scalar_t loss_tmp = conn_ij * p_ij * (diff_norm0*diff_norm0 + diff_norm1*diff_norm1 + diff_norm2*diff_norm2);
       loss_ += loss_tmp;

    }
  }

  loss[ind1] = loss_;

  __syncthreads();
 
}


/**
 * calculate the gradient back-propagated to the offset
 * 	offset 		input, the vertex displacement field of the full grid
 * 	topology	input, the topology probability
 * 	grad_offset	output, gradient on the offset
 * 	mask 		input, mask denoting if two topogolies have connected triangles or not	
 * 	direction	input, a integer denoting the neighoring relationship between two cells 
 * 				0: two cells adajecent in x direction
 * 				1: two cells adajecent in y direction
 * 				2: two cells adajecent in z direction
 * 				3: dummy label for inner cell loss
 */
 template <typename scalar_t>
__global__ void pairwise_grad(const scalar_t *offset, const scalar_t *topology, scalar_t *grad_offset, const scalar_t *mask, const int direction){

  int i1 = blockIdx.x;
  int j1 = blockIdx.y;
  int k1 = threadIdx.x;

  int W = gridDim.x;
  int H = gridDim.y;
  int D = blockDim.x;

  // TODO: change below
  int T = NumTri;
  
  int i2=0, j2=0, k2=0, ind1=0, ind2=0;
  // x direction
  if (direction==0){
       if (i1==W-1) return;
       ind1 = i1*H*D + j1*H + k1;
       ind2 = ind1+H*D;
       i2 = i1+1;
       j2 = j1;
       k2 = k1;
  }
  // y direction
  else if (direction==1){
       if (j1==H-1) return;
       ind1 = i1*H*D + j1*H + k1;
       ind2 = ind1+H;
       i2 = i1;
       j2 = j1+1;
       k2 = k1;
  }
  // z direction
  else if (direction==2){
       if (k1==D-1) return;
       ind1 = i1*H*D + j1*H + k1;
       ind2 = ind1+1;
       i2 = i1;
       j2 = j1;
       k2 = k1+1;
  }
  // inner loss, within the same cell
  else if (direction==3){
       ind1 = i1*H*D + j1*H + k1;
       ind2 = ind1;
       i2 = i1;
       j2 = j1;
       k2 = k1;
  }
  
  // get normal vector in both grids
  scalar_t norm1[NumTri*3];
  scalar_t norm2[NumTri*3];
  scalar_t length1[NumTri];
  scalar_t length2[NumTri];
  offset_to_normals(offset, W+1, H+1, D+1, i1, j1, k1, direction*2, norm1, length1);
  offset_to_normals(offset, W+1, H+1, D+1, i2, j2, k2, direction*2+1, norm2, length2);

  scalar_t grad_norm1[NumTri*3]={0};
  scalar_t grad_norm2[NumTri*3]={0};

  for (int ti=0; ti<T; ti++){

    for (int tj=0; tj<T; tj++){

       // no loss if two topologies are not connected
       scalar_t conn_ij = mask[ti*T + tj];
       if (conn_ij<eps) continue;

       // joint probability of two topology combinations
       // corresponding to outer product
       scalar_t pi = topology[ind1*T+ti];
       scalar_t pj = topology[ind2*T+tj];
       scalar_t p_ij = pi*pj;

       // l2 loss
       scalar_t grad_norm1_[3] = {-2*conn_ij*p_ij*norm2[tj*3 + 0], -2*conn_ij*p_ij*norm2[tj*3 + 1], -2*conn_ij*p_ij*norm2[tj*3 + 2]};
       scalar_t grad_norm2_[3] = {-2*conn_ij*p_ij*norm1[ti*3 + 0], -2*conn_ij*p_ij*norm1[ti*3 + 1], -2*conn_ij*p_ij*norm1[ti*3 + 2]};

       scalar_t norm1_[3] = {norm1[ti*3 + 0], norm1[ti*3 + 1], norm1[ti*3 + 2]};
       scalar_t norm2_[3] = {norm2[tj*3 + 0], norm2[tj*3 + 1], norm2[tj*3 + 2]};
       grad_normalized_to_normal(grad_norm1_, norm1_, length1[ti]);
       grad_normalized_to_normal(grad_norm2_, norm2_, length2[tj]);
       grad_norm1[ti*3 + 0] += grad_norm1_[0]; 
       grad_norm1[ti*3 + 1] += grad_norm1_[1];
       grad_norm1[ti*3 + 2] += grad_norm1_[2];

       grad_norm2[tj*3 + 0] += grad_norm2_[0];
       grad_norm2[tj*3 + 1] += grad_norm2_[1];
       grad_norm2[tj*3 + 2] += grad_norm2_[2];

    }
  }

  grad_normal_to_offset(grad_offset, grad_norm1, offset, W+1, H+1, D+1, i1, j1, k1, direction*2);
  grad_normal_to_offset(grad_offset, grad_norm2, offset, W+1, H+1, D+1, i2, j2, k2, direction*2 + 1);

}

} //namespace




/*
 * Forward function, calculating the distance from a set of points to one single linesegment 
 * params: 
 *     offset 		input, vertex displacement field, 3x(W+1)x(H+1)x(D+1) 
 * 	   topolopy 	input, probability for each topology, (WxHxD)xT', T' is the number of triangles instead of topologies 
 *     xTable	   	input, connected triangles in x direction, T'xT' 
 *  	 yTable	 	  input, connected triangles in y direction, T'xT'
 *     zTable	 	  input, connected triangles in z direction, T'xT'
 *  	 innerTable	input, connected triangles within the same topology, T'xT'
 *     loss   		output, smoothness loss
 */	
void curvature_constraint_kernel_forward( 
    at::Tensor offset,
    at::Tensor topology,
    at::Tensor xTable,
    at::Tensor yTable,
    at::Tensor zTable,
    at::Tensor innerTable,
    at::Tensor loss_x,
    at::Tensor loss_y,
    at::Tensor loss_z,
    at::Tensor loss_inner){
  
  int W = offset.size(1)-1;
  int H = offset.size(2)-1;
  int D = offset.size(3)-1;
  dim3 dimGrid(W, H, 1);
  dim3 dimBlock(D, 1, 1);


  // Some checks
  assert(offset.type().scalarType() == at::ScalarType::Float);
  assert(topology.type().scalarType() == at::ScalarType::Float);
  assert(xTable.type().scalarType() == at::ScalarType::Float);
  assert(yTable.type().scalarType() == at::ScalarType::Float);
  assert(zTable.type().scalarType() == at::ScalarType::Float);
  assert(innerTable.type().scalarType() == at::ScalarType::Float);
  assert(loss_x.type().scalarType() == at::ScalarType::Float);
  assert(loss_y.type().scalarType() == at::ScalarType::Float);
  assert(loss_z.type().scalarType() == at::ScalarType::Float);
  assert(loss_inner.type().scalarType() == at::ScalarType::Float);

  // Lauch the kernels
  // x loss
  pairwise_loss<float><<< dimGrid, dimBlock>>>(
      offset.data<float>(),
      topology.data<float>(),
      xTable.data<float>(),
      loss_x.data<float>(),
      0);
  // y loss
  pairwise_loss<float><<< dimGrid, dimBlock>>>(
      offset.data<float>(),
      topology.data<float>(),
      yTable.data<float>(),
      loss_y.data<float>(),
      1);
  // z loss
  pairwise_loss<float><<< dimGrid, dimBlock>>>(
      offset.data<float>(),
      topology.data<float>(),
      zTable.data<float>(),
      loss_z.data<float>(),
      2);
  // inner loss
  pairwise_loss<float><<< dimGrid, dimBlock>>>(
      offset.data<float>(),
      topology.data<float>(),
      innerTable.data<float>(),
      loss_inner.data<float>(),
      3);
}

/*
 * Backward function, calculating the derivative of the topology with respect to the loss 
 * params: 
 * 	  grad_output   input, gradient on the output loss, 1
 *	  offset 		    input, vertex displacement field, 3x(W+1)x(H+1)x(D+1)
 * 	  topolopy 		  input, probability for each topology, (WxHxD)xT', T' is the number of triangles instead of topologies 
 *  	xTable	 	    input, connected triangles in x direction, T'xT' 
 *  	yTable	 	    input, connected triangles in y direction, T'xT'
 *  	zTable	 	    input, connected triangles in z direction, T'xT'
 *  	innerTable	 	input, connected triangles within the same topology, T'xT'
 *  	grad_offset  	output, gradient on the offset, 3x(W+1)x(H+1)x(D+1)
 *
 */	
void curvature_constraint_kernel_backward(
    at::Tensor grad_output, 
    at::Tensor offset,
    at::Tensor topology,
    at::Tensor xTable,
    at::Tensor yTable,
    at::Tensor zTable,
    at::Tensor innerTable,
    at::Tensor grad_offset){
  int W = offset.size(1) - 1;
  int H = offset.size(2) - 1;
  int D = offset.size(3) - 1;

  dim3 dimGrid(W, H, 1);
  dim3 dimBlock(D, 1, 1);

  assert(offset.type().scalarType() == at::ScalarType::Float);
  assert(topology.type().scalarType() == at::ScalarType::Float);
  assert(xTable.type().scalarType() == at::ScalarType::Float);
  assert(yTable.type().scalarType() == at::ScalarType::Float);
  assert(zTable.type().scalarType() == at::ScalarType::Float);
  assert(innerTable.type().scalarType() == at::ScalarType::Float);
  assert(grad_output.type().scalarType() == at::ScalarType::Float);
  assert(grad_offset.type().scalarType() == at::ScalarType::Float);
  
  // lauch the kernel
  pairwise_grad<float><<< dimGrid, dimBlock>>>(
    offset.data<float>(),
    topology.data<float>(),
    grad_offset.data<float>(),
    xTable.data<float>(),
    0);

  pairwise_grad<float><<< dimGrid, dimBlock>>>(
    offset.data<float>(),
    topology.data<float>(),
    grad_offset.data<float>(),
    yTable.data<float>(),
    1);

  pairwise_grad<float><<< dimGrid, dimBlock>>>(
    offset.data<float>(),
    topology.data<float>(),
    grad_offset.data<float>(),
    zTable.data<float>(),
    2);

  pairwise_grad<float><<< dimGrid, dimBlock>>>(
    offset.data<float>(),
    topology.data<float>(),
    grad_offset.data<float>(),
    innerTable.data<float>(),
    3);

  // Multiply with incoming gradient
  // Do that in Python now
  // grad_offset *= grad_output;
}
  

